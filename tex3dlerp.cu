#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>
#include <stdexcept>
#include <string>
#include <memory>
#include <vector>
#include <cassert>
#include <iostream>

static const char *_cudaGetErrorEnum(hipError_t error) {
  return hipGetErrorName(error);
}

template <typename T>
void check(T result, char const *const func, const char *const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
            static_cast<unsigned int>(result), _cudaGetErrorEnum(result), func);
    exit(EXIT_FAILURE);
  }
}

inline void __getLastCudaError(const char *errorMessage, const char *file,
                               const int line) {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr,
            "%s(%i) : getLastCudaError() CUDA error :"
            " %s : (%d) %s.\n",
            file, line, errorMessage, static_cast<int>(err),
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

#define getLastCudaError(msg) __getLastCudaError(msg, __FILE__, __LINE__)

typedef unsigned int uint;
typedef unsigned short ushort;
typedef unsigned char uchar;


template <typename T = float>
class CUDATexture3D {
    static_assert(std::is_arithmetic_v<T>, "CUDATexture3D only supports arithmetic types.");
public:
    using Ptr = std::shared_ptr<CUDATexture3D>;

    // Constructor: Wraps the volume object and device buffer
    CUDATexture3D(size_t width, size_t height, size_t depth, const void* volumeData, 
        bool normalizedCoords, 
        hipTextureFilterMode filter = hipFilterModeLinear, 
        hipTextureAddressMode addressMode = hipAddressModeClamp)
        : width(width), height(height), depth(depth) 
    {
        // Create channel description for the template type T
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();
        hipExtent extent = make_hipExtent(width, height, depth);

        // Allocate 3D array
        if (hipMalloc3DArray(&hipArray, &channelDesc, extent) != hipSuccess) {
            throw std::runtime_error("Failed to allocate CUDA 3D array.");
        }

        // Copy data to the CUDA 3D array
        hipMemcpy3DParms copyParams = {};
        copyParams.srcPtr = make_hipPitchedPtr(const_cast<void*>(volumeData), width * sizeof(T), width, height);
        copyParams.dstArray = hipArray;
        copyParams.extent = extent;
        copyParams.kind = hipMemcpyHostToDevice;

        if (hipMemcpy3D(&copyParams) != hipSuccess) {
            hipFreeArray(hipArray);
            throw std::runtime_error("Failed to copy data to CUDA 3D array.");
        }

        // Create texture object
        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = hipArray;

        hipTextureDesc texDesc = {};
        texDesc.addressMode[0] = addressMode;
        texDesc.addressMode[1] = addressMode;
        texDesc.addressMode[2] = addressMode;
        texDesc.filterMode = filter;
        texDesc.readMode = hipReadModeElementType;
        texDesc.normalizedCoords = normalizedCoords;

        if (hipCreateTextureObject(&textureObject, &resDesc, &texDesc, nullptr) != hipSuccess) {
            hipFreeArray(hipArray);
            throw std::runtime_error("Failed to create CUDA texture object.");
        }
    }

    // Destructor: Ensures resources are freed
    ~CUDATexture3D() {
        if (textureObject) {
            hipDestroyTextureObject(textureObject);
        }
        if (hipArray) {
            hipFreeArray(hipArray);
        }
    }

    // Delete copy constructor and copy assignment
    CUDATexture3D(const CUDATexture3D&) = delete;
    CUDATexture3D& operator=(const CUDATexture3D&) = delete;

    // Allow move constructor and move assignment
    CUDATexture3D(CUDATexture3D&& other) noexcept
        : hipArray(other.hipArray), textureObject(other.textureObject),
          width(other.width), height(other.height), depth(other.depth) 
    {
        other.hipArray = nullptr;
        other.textureObject = 0;
    }

    CUDATexture3D& operator=(CUDATexture3D&& other) noexcept {
        if (this != &other) {
            // Clean up current resources
            if (textureObject) hipDestroyTextureObject(textureObject);
            if (hipArray) hipFreeArray(hipArray);

            // Transfer ownership
            hipArray = other.hipArray;
            textureObject = other.textureObject;
            width = other.width;
            height = other.height;
            depth = other.depth;

            other.hipArray = nullptr;
            other.textureObject = 0;
        }
        return *this;
    }

    // Accessor for texture object
    hipTextureObject_t getTextureObject() const { return textureObject; }

private:
    hipArray_t hipArray = nullptr;
    hipTextureObject_t textureObject = 0;
    size_t width = 0, height = 0, depth = 0;

};


__device__ float LinearSampleVolume(hipTextureObject_t volumeTex, float3 texCoord) {
    float v = tex3D<float>(volumeTex, texCoord.x, texCoord.y, texCoord.z);
    return v;
}

// Kernel to sample and print node values
//              width = 4
//              |-----o-----|-----o-----|-----o-----|-----o-----|
//      node    0           1           2           3           4   node count = 5, node max index = 4      // unnormalized coord
//      cell          0           1           2           3         cell count = 4, cell max index = 3      // normalied coord count
//      sig            -----------  ---------   ----------
//      seg                0            1           2               seg count = 3,  seg max index 2         // normalized corrd segment
//
//                    |                                    |
//      norm    0   0.125       0.325      0.625         0.875  1
//      unnorm  0    0.5                                  3.5   4
__global__ void tex1d_sample_kernel(hipTextureObject_t tex, uint3 gridSize, uint3 dispatchSize) {
    // Compute thread-specific coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // if (x >= gridSize.x || y >= gridSize.y || z >= gridSize.z) return;

    float3 coord;
    coord.x = (x * 1.0) / (dispatchSize.x - 1);     // segment normalize 🌙
    // coord.x = coord.x = (x * 1.0) / 4.0;
    coord.y = 0.0;
    coord.z = 0.0;

    float dx = 1.0/ 8.0; //0.125 * 0.5;  🌙❤️ dx must be 2^n or 2^-n

    coord.x *= (gridSize.x - 1);                    // to int coord🌙
    coord.y = 0.0;
    coord.z = 0.0;

    // Unnormalized texture coordinates at texel center
    float3 texCoord = make_float3(coord.x + 0.5, coord.y + 0.5, coord.z + 0.5); // 🌙 int coord 0.5
    float3 offsetCoord = make_float3(coord.x + 0.5 + dx, coord.y, coord.z);

    // Sample at texCoord and offsetCoord
    float nodeValue = LinearSampleVolume(tex, texCoord);
    float offsetValue = LinearSampleVolume(tex, offsetCoord);

    // Print results
    printf("Node (%d, %d, %d) (%f %f %f): Value = %f, (+%f +0 +0) Offset Value = %f dv:%f\n", 
        x, y, z, 
        coord.x, coord.y, coord.z, nodeValue, 
        dx, offsetValue, offsetValue - nodeValue);
}

void launch_test_tex1d_sample() {

    // Define volume size
    constexpr size_t width = 4;
    constexpr size_t height = 1;
    constexpr size_t depth = 1;
    uint3 gridSize = make_uint3(width, height, depth);
    uint3 dispatchSize = make_uint3(width * 3 + 1, height * 1, depth * 1);
    hipExtent volumeSize = make_hipExtent(gridSize.x, gridSize.y, gridSize.z);

    // Allocate and initialize host volume data
    float *h_volume = new float[gridSize.x * gridSize.y * gridSize.z];
    auto h_grid = reinterpret_cast<float(*)[height][depth]>(h_volume);
    for (size_t z = 0; z < depth; z++) {
        for (size_t y = 0; y < height; y++) {
            for (size_t x = 0; x < width; x++) {
                h_grid[z][y][x] = x + y + z;
            }
        }
    }

    // Create texture object
    CUDATexture3D<float> texture(width, height, depth, h_volume, false);
    delete[] h_volume;

    // Launch kernel
    hipTextureObject_t texObj = texture.getTextureObject();
    dim3 blockSize(8, 1, 1);
    dim3 gridSizeBlocks((dispatchSize.x + 7) / 8, (dispatchSize.y + 7) / 8, (dispatchSize.z + 7) / 8);
    tex1d_sample_kernel<<<gridSizeBlocks, blockSize>>>(texObj, gridSize, dispatchSize);
    hipDeviceSynchronize();
}


// Kernel to sample and print node values
__global__ void tex1d_sample_norm_kernel(hipTextureObject_t tex, uint3 gridSize, uint3 dispatchSize) {
    // Compute thread-specific coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;

    // if (x >= gridSize.x || y >= gridSize.y || z >= gridSize.z) return;

    float3 coord;
    coord.x = (x * 1.0) / (dispatchSize.x - 1);
    // coord.x = coord.x = (x * 1.0) / 4.0;
    coord.y = 0.0;
    coord.z = 0.0;

    coord.x = (coord.x * (gridSize.x - 1) + 0.5) / gridSize.x;
    // coord.x *= (gridSize.x - 1);
    // coord.y = 0.0;
    // coord.z = 0.0;

    // coord.x = 0.25 + 0.125;

    // Unnormalized texture coordinates at texel center
    float3 texCoord = make_float3(coord.x, coord.y, coord.z);
    float dx = 1.0/ 32.0; //0.125 * 0.5;  🌙❤️ dx must be 2^n or 2^-n
    float3 offsetCoord = make_float3(coord.x + dx / gridSize.x, coord.y, coord.z);

    // Sample at texCoord and offsetCoord
    float nodeValue = LinearSampleVolume(tex, texCoord);
    float offsetValue = LinearSampleVolume(tex, offsetCoord);

    // Print results
    printf("Node (%d, %d, %d) (%f %f %f): Value = %f, (+%f +0 +0) Offset Value = %f dv:%f\n", x, y, z, 
        coord.x, coord.y, coord.z, nodeValue, 
        dx, offsetValue, offsetValue - nodeValue);
}

void launch_test_tex1d_sample_norm() {

    // Define volume size
    constexpr size_t width = 4;
    constexpr size_t height = 1;
    constexpr size_t depth = 1;
    uint3 gridSize = make_uint3(width, height, depth);
    uint3 dispatchSize = make_uint3(width * 3 + 1, height * 1, depth * 1);
    hipExtent volumeSize = make_hipExtent(gridSize.x, gridSize.y, gridSize.z);

    // Allocate and initialize host volume data
    float *h_volume = new float[gridSize.x * gridSize.y * gridSize.z];
    auto h_grid = reinterpret_cast<float(*)[height][depth]>(h_volume);
    for (size_t z = 0; z < depth; z++) {
        for (size_t y = 0; y < height; y++) {
            for (size_t x = 0; x < width; x++) {
                h_grid[z][y][x] = x + y + z;
            }
        }
    }

    // Create texture object
    CUDATexture3D<float> texture(width, height, depth, h_volume, true);
    delete[] h_volume;

    // Launch kernel
    hipTextureObject_t texObj = texture.getTextureObject();
    dim3 blockSize(8, 1, 1);
    dim3 gridSizeBlocks((dispatchSize.x + 7) / 8, (dispatchSize.y + 7) / 8, (dispatchSize.z + 7) / 8);
    tex1d_sample_norm_kernel<<<gridSizeBlocks, blockSize>>>(texObj, gridSize, dispatchSize);
    hipDeviceSynchronize();
}


int main() {
    std::cout << "Hello from C++!" << std::endl;
    
    launch_test_tex1d_sample();
    // launch_test_tex1d_sample_norm();

    return 0;
}
